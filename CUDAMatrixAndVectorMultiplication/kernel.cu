#include "hip/hip_runtime.h"
#include "kernel.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>

__global__ void function(float* dA, float* dX, float* dC, int vectorSize)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < vectorSize) {
        float sum = 0.0f;
        for (int j = 0; j < vectorSize; j++) {
            sum += dA[i * vectorSize + j] * dX[j];
        }
        dC[i] = sum;
    }

}