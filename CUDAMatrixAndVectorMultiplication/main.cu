#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "kernel.h"

int main(int argc, char* argv[])
{

	float* hA;
	float* dA;

	float* hX;
	float* dX;

	float* hC;
	float* dC;

	int  N_thread = 32;
	int vectorSize = N_thread *1;
	int matrixSize = vectorSize * vectorSize;
	int N_blocks;
	int i;
	int j;
	unsigned int vectorMem_size = sizeof(float) * vectorSize;
	unsigned int matrixMem_size = sizeof(float) * matrixSize;

	hA = (float*)malloc(matrixMem_size);
	hX = (float*)malloc(vectorMem_size);
	hC = (float*)malloc(vectorMem_size);

	hipError_t err;

	err = hipMalloc((void**)&dA, matrixMem_size);
	if (err != hipSuccess) {
		fprintf(stderr, "Cannot allocate GPU memory: %s\n", hipGetErrorString(err));
		return 1;
	}

	err = hipMalloc((void**)&dX, vectorMem_size);
	if (err != hipSuccess) {
		fprintf(stderr, "Cannot allocate GPU memory: %s\n", hipGetErrorString(err));
		return 1;
	}

	err = hipMalloc((void**)&dC, vectorMem_size);
	if (err != hipSuccess) {
		fprintf(stderr, "Cannot allocate GPU memory: %s\n", hipGetErrorString(err));
		return 1;
	}

	for (i = 0; i < vectorSize; i++) {
		for (int j = 0; j < vectorSize; j++) {
			hA[i * vectorSize + j] = j;
			printf("A[%d,%d] = %.5f\n", i,j, hA[i * vectorSize + j]);
		}
		hX[i] = i;
		hC[i] = 0.0f;
		printf("x[%d] = %.5f\n", i, hX[i]);
	}

	N_blocks = matrixSize / N_thread;

	hipMemcpy(dA , hA , matrixMem_size, hipMemcpyHostToDevice);
	hipMemcpy(dX , hX , vectorMem_size, hipMemcpyHostToDevice);

	function << < N_blocks, N_thread >> > (dA , dX , dC , vectorSize);

	hipMemcpy(hC, dC, vectorMem_size, hipMemcpyDeviceToHost);

	for (int idx = 0; idx < vectorSize; idx++) {
		printf("c[%d] = %.5f\n", idx, hC[idx]);
	}

	free(hA);
	free(hX);
	free(hC);

	hipFree(dA);
	hipFree(dX);
	hipFree(dC);


	return 0;
}